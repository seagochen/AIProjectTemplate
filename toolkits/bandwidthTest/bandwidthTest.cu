#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#ifndef CHECK_CUDA
#define CHECK_CUDA(call)                                                 \
    do {                                                                 \
        hipError_t err = call;                                          \
        if (err != hipSuccess) {                                        \
            fprintf(stderr, "CUDA error in %s (%d): %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    } while (0)
#endif

// 简单的计时函数，用cudaEvent
float measureCopyTime(void* dst, const void* src, size_t size, hipMemcpyKind kind, int repeats=10) {
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // 预热一次
    CHECK_CUDA(hipMemcpy(dst, src, size, kind));
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipEventRecord(start, 0));
    for (int i = 0; i < repeats; i++) {
        CHECK_CUDA(hipMemcpy(dst, src, size, kind));
    }
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    float ms = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));

    // 清理
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    // 返回单次拷贝时间，单位：毫秒
    return ms / repeats;
}

int main(int argc, char** argv) {
    // 测试数据大小(字节)
    // 可以自行调整，比如 100 MB = 100 * 1024 * 1024 bytes
    size_t dataSize = 100 * 1024 * 1024; 

    // 分配主机内存
    void* h_src = malloc(dataSize);
    void* h_dst = malloc(dataSize);
    if (!h_src || !h_dst) {
        fprintf(stderr, "Host memory allocation failed\n");
        return EXIT_FAILURE;
    }

    // 初始化主机数据
    memset(h_src, 1, dataSize);
    memset(h_dst, 0, dataSize);

    // 分配设备内存
    void* d_src;
    void* d_dst;
    CHECK_CUDA(hipMalloc(&d_src, dataSize));
    CHECK_CUDA(hipMalloc(&d_dst, dataSize));

    // ===========================
    // Host to Device 带宽测试
    // ===========================
    float h2d_time = measureCopyTime(d_dst, h_src, dataSize, hipMemcpyHostToDevice);
    float h2d_bandwidth = (dataSize * 1e-9f) / (h2d_time * 1e-3f); // GB/s
    printf("Host to Device: %.3f GB/s\n", h2d_bandwidth);

    // ===========================
    // Device to Host 带宽测试
    // ===========================
    float d2h_time = measureCopyTime(h_dst, d_dst, dataSize, hipMemcpyDeviceToHost);
    float d2h_bandwidth = (dataSize * 1e-9f) / (d2h_time * 1e-3f); // GB/s
    printf("Device to Host: %.3f GB/s\n", d2h_bandwidth);

    // ===========================
    // Device to Device 带宽测试
    // ===========================
    float d2d_time = measureCopyTime(d_dst, d_src, dataSize, hipMemcpyDeviceToDevice);
    float d2d_bandwidth = (dataSize * 1e-9f) / (d2d_time * 1e-3f); // GB/s
    printf("Device to Device: %.3f GB/s\n", d2d_bandwidth);

    // 清理内存
    free(h_src);
    free(h_dst);
    CHECK_CUDA(hipFree(d_src));
    CHECK_CUDA(hipFree(d_dst));

    return 0;
}
